#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector_functions.h>
#include <math.h>

#include <vector>

#define SQRT2 1.414213562373095
#define MIN -0.751791631228899

namespace {

template <typename scalar_t>
__device__ __forceinline__ float gelu_backward(float g, scalar_t o, bool m) {
    float rval;
    // Polynomial approximation of different regimes - no closed-form solution (transcendental)
    // Splined Polynomial see splrep and PPoly from scipy
    // Horner's method do polynomial evalutation in O(n)
    // 2 Cases, before and after x-minimum for inverse
    if (m) {
      if (o > ((float) 0.97742819)) { // (0.97742819, inf)
        if (o < ((float) 3.0)) { // (0.97742819, 3.0)
          float o_translated = o + ((float) 0.9774323423471688);
          rval = 
            o_translated * 
            (((float) 1.40963829e-01) + o_translated * 
            (((float) -2.74712808e-01) + o_translated * 
            (((float) 1.30758747e-01) + o_translated * 
            (((float) -3.34020163e-02) + o_translated * 
            (((float) 2.21484679e-02) + o_translated * 
            (((float) -1.73618287e-02) + o_translated * 
            (((float) 1.13268204e-02) + o_translated * 
            (((float) -7.30845601e-03) + o_translated * 
            (((float) 3.40900094e-03) + o_translated * 
            (((float) -9.19837579e-04) + o_translated * 
            (((float) 1.24505621e-04) + o_translated * ((float) -6.11869959e-06) 
            ))))))))))) + ((float) 1.1079279405461824);
        } else { // [3.0, inf)
          rval = 1.0;
        }
      } else { // (-0.16997121, 0.97742819)
        if (o < ((float) 0.10682432)) { // (-0.16997121, 0.10682432)
          float o_translated = o + ((float) 0.1699712074799012);
          rval = 
            o_translated * 
            (((float) 2.03860722e+01) + o_translated * 
            (((float) -1.51033017e+03) + o_translated * 
            (((float) 7.29893892e+04) + o_translated * 
            (((float) -2.08603338e+06) + o_translated * 
            (((float) 3.75106859e+07) + o_translated * 
            (((float) -4.43847941e+08) + o_translated * 
            (((float) 3.54134638e+09) + o_translated * 
            (((float) -1.91638067e+10) + o_translated * 
            (((float) 6.93203478e+10) + o_translated * 
            (((float) -1.60441793e+11) + o_translated * 
            (((float) 2.14787711e+11) + o_translated * ((float) -1.26477496e+11) 
            )))))))))));
        } else { // (0.10682432, 0.97742819)
          float o_translated = o - ((float) 0.10682674544528972);
          rval = 
            o_translated * 
            (((float) 1.19132403) + o_translated * 
            (((float) -1.44264915) + o_translated * 
            (((float) 1.9529612) + o_translated * 
            (((float) -3.84418714) + o_translated * 
            (((float) 8.2541642) + o_translated * 
            (((float) -16.40234298) + o_translated * 
            (((float) 27.17877296) + o_translated * 
            (((float) -34.79939566) + o_translated * 
            (((float) 32.32044636) + o_translated * 
            (((float) -20.25396347) + o_translated * 
            (((float) 7.6092532) + o_translated * ((float) -1.28943268) 
            ))))))))))) + ((float) 0.6468323960187936);
        }
      }
    } else {
        if (o > ((float) -3.0)) {
          float o_translated = o + ((float) 0.1699712074799012);
          rval = 
            o_translated * 
            (((float) -2.44705742e+01) + o_translated * 
            (((float) 3.64509157e+03) + o_translated * 
            (((float) -3.24039623e+05) + o_translated * 
            (((float) 1.72624707e+07) + o_translated * 
            (((float) -5.87053927e+08) + o_translated * 
            (((float) 1.33643456e+10) + o_translated * 
            (((float) -2.09622261e+11) + o_translated * 
            (((float) 2.29518632e+12) + o_translated * 
            (((float) -1.75056809e+13) + o_translated * 
            (((float) 9.11525632e+13) + o_translated * 
            (((float) -3.08903527e+14) + o_translated * 
            (((float) 6.14002866e+14) + o_translated * ((float) -5.43092154e+14) 
            ))))))))))));
        } else {
          rval = 0.0;
        }
    }
    return rval * g;
}

template <typename scalar_t>
__global__ void gelu_cuda_backward_kernel(
    size_t size,
    const float4* __restrict__ grad_output,
    const float4* __restrict__ output,
    const uchar4* __restrict__ mask,
    float4* grad_input) {
    // Calculate grad_input using output and grad_output only. Uses mask for invertibility and polynomial approximations
    #pragma unroll
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < size; 
         i += blockDim.x * gridDim.x){
        float4 go = grad_output[i];
        float4 o = output[i];
        uchar4 m = mask[i];
        float4 gi;
        gi = make_float4(gelu_backward<scalar_t>(go.x, o.x, (bool) m.x),
                         gelu_backward<scalar_t>(go.y, o.y, (bool) m.y),
                         gelu_backward<scalar_t>(go.z, o.z, (bool) m.z),
                         gelu_backward<scalar_t>(go.w, o.w, (bool) m.w));
        grad_input[i] = gi;
  }
}


template <typename scalar_t>
__device__ __forceinline__ float gelu_forward(float x) {
    /* GELU Hendrycks */
    float rval;
    rval = ((float) 0.5) * x * (((float) 1.0) + erf(x/((float) SQRT2)));
    return rval; 
}

template <typename scalar_t>
__global__ void gelu_cuda_forward_kernel(
    size_t size,
    const float4* __restrict__ input,
    float4* output,
    uchar4* mask) {
    #pragma unroll
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
         i < size; 
         i += blockDim.x * gridDim.x){
        float4 inp = input[i];
        float4 out;
        uchar4 m;
        out = make_float4(gelu_forward<scalar_t>(inp.x),
                          gelu_forward<scalar_t>(inp.y),
                          gelu_forward<scalar_t>(inp.z),
		 	  gelu_forward<scalar_t>(inp.w));

	m = make_uchar4((inp.x >= ((float) MIN)) ? 255 : 0,
                        (inp.y >= ((float) MIN)) ? 255 : 0,
                        (inp.z >= ((float) MIN)) ? 255 : 0,
                        (inp.w >= ((float) MIN)) ? 255 : 0);

	output[i] = out;
	mask[i] = m;
  }
}

} // namespace

std::vector<torch::Tensor> gelu_cuda_forward(
    torch::Tensor input) {

  auto output_options =
    torch::TensorOptions()
    .dtype(torch::kFloat32)
    .layout(torch::kStrided)
    .device(torch::kCUDA)
    .requires_grad(false);

  auto mask_options =
    torch::TensorOptions()
    .dtype(torch::kBool)
    .layout(torch::kStrided)
    .device(torch::kCUDA)
    .requires_grad(false);

  auto output = torch::empty_like(input, output_options);
  auto mask = torch::empty_like(input, mask_options);

  size_t nelement = torch::numel(input);
  size_t thread_size = 64;

  const dim3 threads(thread_size);
  const dim3 blocks(((nelement / 4) + thread_size - 1) / thread_size); // Ceil Trick: https://stackoverflow.com/questions/62032583/division-round-up-in-c

  AT_DISPATCH_FLOATING_TYPES(input.scalar_type(), "gelu_cuda_forward", ([&] {
    gelu_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
        nelement/4,
        (float4*)input.data_ptr(),
        (float4*)output.data_ptr(),
        (uchar4*)mask.data_ptr());
  }));

  return {output, mask};
}

std::vector<torch::Tensor> gelu_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor output,
    torch::Tensor mask) {
  
  // Create output tensor

  auto grad_input_options =
    torch::TensorOptions()
    .dtype(torch::kFloat32)
    .layout(torch::kStrided)
    .device(torch::kCUDA)
    .requires_grad(false);

  auto grad_input = torch::empty_like(output, grad_input_options);

  size_t nelement = torch::numel(output);
  size_t thread_size = 64;

  const dim3 threads(thread_size);
  const dim3 blocks(((nelement / 4) + thread_size - 1) / thread_size); // Ceil Trick: https://stackoverflow.com/questions/62032583/division-round-up-in-c

  AT_DISPATCH_FLOATING_TYPES(output.scalar_type(), "gelu_cuda_backward", ([&] {
    gelu_cuda_backward_kernel<scalar_t><<<blocks, threads>>>(
        nelement/4,
        (float4*)grad_output.data_ptr(),
        (float4*)output.data_ptr(),
        (uchar4*)mask.data_ptr(),
        (float4*)grad_input.data_ptr());
  }));

  return {grad_input};
}
